#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <hip/hip_runtime.h>

__global__ void test1_kernel(int* result) {
    *result = 1000;  // CUDA kernel sets value to 0
}

int test1() {
    int* d_result;
    int h_result;

    hipMalloc(&d_result, sizeof(int));
    test1_kernel<<<1, 1>>>(d_result);
    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_result);

    return h_result;
}

PYBIND11_MODULE(naive, m) {
    m.def("test1", &test1, "A CUDA function that returns 0");
}
